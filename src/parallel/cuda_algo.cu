#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_WIDTH 16

__global__ void matrixMulKernel(float* d_A, float* d_B, float* d_C, int numARows, int numACols, int numBCols) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Cvalue = 0.0;

    for (int t = 0; t < (numACols - 1) / TILE_WIDTH + 1; ++t) {
        if (Row < numARows && t * TILE_WIDTH + tx < numACols)
            ds_A[ty][tx] = d_A[Row * numACols + t * TILE_WIDTH + tx];
        else
            ds_A[ty][tx] = 0.0;

        if (Col < numBCols && t * TILE_WIDTH + ty < numACols)
            ds_B[ty][tx] = d_B[(t * TILE_WIDTH + ty) * numBCols + Col];
        else
            ds_B[ty][tx] = 0.0;

        __syncthreads();

        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; ++i)
            Cvalue += ds_A[ty][i] * ds_B[i][tx];

        __syncthreads();
    }

    if (Row < numARows && Col < numBCols)
        d_C[Row * numBCols + Col] = Cvalue;
}

void matrixMultiply(float* h_A, float* h_B, float* h_C, int numARows, int numACols, int numBCols) {
    float *d_A, *d_B, *d_C;
    size_t sizeA = numARows * numACols * sizeof(float);
    size_t sizeB = numACols * numBCols * sizeof(float);
    size_t sizeC = numARows * numBCols * sizeof(float);

    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((numBCols - 1) / TILE_WIDTH + 1, (numARows - 1) / TILE_WIDTH + 1);

    matrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, numARows, numACols, numBCols);

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int numARows = 4, numACols = 4, numBCols = 4;
    float h_A[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    float h_B[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    float h_C[numARows * numBCols];

    matrixMultiply(h_A, h_B, h_C, numARows, numACols, numBCols);

    for (int i = 0; i < numARows; ++i) {
        for (int j = 0; j < numBCols; ++j) {
            std::cout << h_C[i * numBCols + j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}

// You are given an array of floats of size N. Your task is to write a CUDA program to find the sum of all elements in this array.

#define BLOCK_SIZE 256

__global__ void sharedMemReduction(float *input, float *output, int N) {
    __shared__ float sharedData[BLOCK_SIZE];

    int tid = threadIdx.x;
    int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;

    sharedData[tid] = (globalIndex < N) ? input[globalIndex] : 0.0f;
    __syncthreads();

    // Perform Reduction in Shared Memory
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }

    // Write block result to global memory
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

void sumReduction(float *h_input, int N) {
    float *d_input, *d_output;
    int gridSize = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, gridSize * sizeof(float));

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    sharedMemReduction<<<gridSize, BLOCK_SIZE>>>(d_input, d_output, N);

    // Copy partial results back to host
    float *h_partialSums = new float[gridSize];
    hipMemcpy(h_partialSums, d_output, gridSize * sizeof(float), hipMemcpyDeviceToHost);

    // Final sum on CPU
    float totalSum = 0;
    for (int i = 0; i < gridSize; i++) {
        totalSum += h_partialSums[i];
    }

    std::cout << "Final Sum: " << totalSum << std::endl;

    delete[] h_partialSums;
    hipFree(d_input);
    hipFree(d_output);
}

